
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

__host__ void HostOnly() {
  std::cout << "This function may only be called from host" << std::endl;
}

__device__ void DeviceOnly() {
  printf("This function may only be called from device\n");
}

__host__ __device__ void HostDevicePrint() {
#if defined(__CUDA_ARCH__)
  printf("host & device print\n");
#else
  std::cout << "host & device print" << std::endl;
#endif
}

__host__ __device__ float SquareAnywhere(float x)
{
    return x * x;
}

__global__ void RunGPU(float x)
{
    DeviceOnly();
    HostDevicePrint();
    printf("%f\n", SquareAnywhere(x));
}

void RunCPU(float x)
{
    HostOnly();
    HostDevicePrint();
    std::cout << SquareAnywhere(x) << std::endl;
}

int main()
{
    std::cout << "==== Sample 02 - Host / Device Functions ====\n" << std::endl;
    /*
     Expected output:
     "This function may only be called from the host"
     1764
     "This function may only be called from the device"
     1764.00
    */

    RunCPU(42);
    RunGPU<<<1, 1>>>(42);
    hipDeviceSynchronize();
    return 0;
}
